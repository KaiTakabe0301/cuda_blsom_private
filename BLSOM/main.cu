#include<iostream>
#include"BLSOM.h"
#include"SelectGPU.h"
#include"LoadDataSet.h"
#include<hiprand/hiprand_kernel.h>


#define MAP_WIDTH 200
#define MAP_HEIGHT 50
#define TRAIN_NUM 200
#define EPOC_NUM 0

int WriteSOMMAP(std::string fileName, float* map, int map_vec, int map_width, int map_height) {
	std::ofstream ofs;
	ofs.open(fileName, 'w');

	if (!ofs) {
		std::cerr << "can't opne file" << std::endl;
		return EXIT_FAILURE;
	}

	ofs << map_vec << std::endl;
	ofs << map_width << std::endl;
	ofs << map_height << std::endl;

	for (int i = 1; i < map_height*map_width; i++) {
		for (int v = 0; v < map_vec; v++) {
			ofs << *map << " ";
			map++;
		}
		ofs << "\n";
	}
	ofs.close();

	return EXIT_SUCCESS;
}

int main(int argc, char** argv) {
	int device;
	int vec_dim;
	int map_width;
	int map_height;
	float* som;
	std::shared_ptr<float> map_weight;
	std::vector<float> trains;
	std::vector<float> ave_vec;
	std::vector<std::vector<float>> rotation;
	std::vector<float> sdev;

	std::vector<std::vector<int>> container;
	std::vector<int> data;

	std::vector<thrust::device_vector<float>> d_con;
	thrust::device_vector<float> d_data;

	for (int i = 1; i <= 9; i++) {
		data.push_back(i);
		d_data.push_back(i);
		if (i % 3 == 0) {
			container.push_back(data);
			data.clear();

			d_con.push_back(d_data);
			d_data.clear();
		}
	}

	for (int i = 0; i <3; i++) {
		for (int j = 0; j < 3; j++) {
			std::cout << &(container[i][j]) << " ";
		}
		std::cout << "\n";
	}
	std::cout << "\n";

	for (int i = 0; i <3; i++) {
		for (int j = 0; j < 3; j++) {
			std::cout << d_con[i][j] << " ";
		}
		std::cout << "\n";
	}


	/* load init data */
	trains = LoadTrain("C:\\Users\\Kai\\Desktop\\mori_PCA\\No1.epc", '\t');
	ave_vec = LoadAverageVector("C:\\Users\\Kai\\Desktop\\mori_PCA\\vector_Ave.txt");
	rotation = LoadRotation("C:\\Users\\Kai\\Desktop\\mori_PCA\\rotation.txt");
	sdev = LoadStandardDev("C:\\Users\\Kai\\Desktop\\mori_PCA\\sdev.txt");

	map_width = MAP_WIDTH;
	map_height = MAP_HEIGHT;
	vec_dim = ave_vec.size();

	BLSOM test = BLSOM(vec_dim, map_width);
	test.Init(sdev[0], sdev[1], rotation[0].data(), rotation[1].data(), ave_vec.data());
	test.SetTrainingData(trains.data(), trains.size() / ave_vec.size());
	test.InitMapWeight(INIT_BATCH);

	/* Get initial map */
	som = test.GetSOMMap();
	WriteSOMMAP("C:\\Users\\Kai\\Desktop\\mori_PCA\\init_batch_map.txt", som, vec_dim, map_width, test.MapHeight());

	/* Learning */
	test.Learning(50);
	
	/* Get Learned Map */
	som = test.GetSOMMap();
	WriteSOMMAP("C:\\Users\\Kai\\Desktop\\mori_PCA\\result_random_20190324.txt", som, vec_dim, map_width, test.MapHeight());
	
	return 0;
}