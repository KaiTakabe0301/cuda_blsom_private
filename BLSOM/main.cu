#include<iostream>
#include"BLSOM.h"
#include"SelectGPU.h"
#include"LoadDataSet.h"
#include<hiprand/hiprand_kernel.h>
#include<algorithm>

#define MAP_WIDTH 200
#define MAP_HEIGHT 50
#define TRAIN_NUM 200
#define EPOC_NUM 0

int WriteSOMMAP(std::string fileName, float* map, int map_vec, int map_width, int map_height) {
	std::ofstream ofs;
	ofs.open(fileName, 'w');

	if (!ofs) {
		std::cerr << "can't opne file" << std::endl;
		return EXIT_FAILURE;
	}

	ofs << map_vec << std::endl;
	ofs << map_width << std::endl;
	ofs << map_height << std::endl;

	for (int i = 1; i < map_height*map_width; i++) {
		for (int v = 0; v < map_vec; v++) {
			ofs << *map << " ";
			map++;
		}
		ofs << "\n";
	}
	ofs.close();

	return EXIT_SUCCESS;
}

int WriteUmatrix(std::string fileName, std::vector<std::vector<float>> umatrix) {
	std::ofstream ofs;
	ofs.open(fileName, 'w');

	if (!ofs) {
		std::cerr << "can't opne file" << std::endl;
		return EXIT_FAILURE;
	}

	for (int h = 0; h < umatrix.size()-1; h++) {
		for (int w = 0; w < umatrix[0].size()-1; w++) {
			ofs << umatrix[h][w];
			if (w != umatrix[0].size() - 2)
				ofs << "\t";
		}
		if (h != umatrix.size() - 2)
			ofs << "\n";
	}
	ofs.close();

	return EXIT_SUCCESS;
}

int main(int argc, char** argv) {
	int device;
	int vec_dim;
	int map_width;
	int map_height;
	float* som;
	std::vector<std::vector<float>> umatrix;

	std::shared_ptr<float> map_weight;
	std::vector<std::vector<float>> train;
	std::vector<std::vector<std::vector<float>>> epocs;

	std::vector<float> ave_vec;
	std::vector<std::vector<float>> rotation;
	std::vector<float> sdev;

	train = LoadTrains("your train data", 'split');
	ave_vec = LoadAverageVector("average vecter");
	rotation = LoadRotation("rotation");
	sdev = LoadStandardDev("standard dev");


	map_width = MAP_WIDTH;
	map_height = MAP_HEIGHT;
	vec_dim = ave_vec.size();

	BLSOM test = BLSOM(vec_dim, map_width);
	test.Init(sdev[0], sdev[1], rotation[0].data(), rotation[1].data(), ave_vec.data());
	test.SetTrainingData(train);
	test.InitMapWeight(INIT_BATCH);

	/* Get initial map */
	som = test.GetSOMMap();
	WriteSOMMAP("init_batch_map.txt", som, vec_dim, map_width, test.MapHeight());

	/* Get initial umatrix */
	umatrix = test.GetUMatrix();
	WriteUmatrix("init_umatrix.txt", umatrix);


	/* Learning */
	test.Learning(50);

	/* Get Learned Map */
	som = test.GetSOMMap();
	WriteSOMMAP("result_batch.txt", som, vec_dim, map_width, test.MapHeight());

	/* Get Umatrix */
	umatrix = test.GetUMatrix();
	WriteUmatrix("learned_umatrix.txt", umatrix);

	return 0;
}